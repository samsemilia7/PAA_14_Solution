#include <stdio.h>
#include <hip/hip_runtime.h>
/* Thread block size = number of threads of a block*/
/* Notice: in this example, the input data size = BLOCK_SIZE */
/*         (different with the CUDA Reduction assigment)     */
#define BLOCK_SIZE 16
__global__ void work_efficient_scan(const float* input, float* output, int size)
{
  /*Declare the shared memory*/
  __shared__ float XY[BLOCK_SIZE];
  /**/
  unsigned int t = threadIdx.x;
  /*load data from global memory to shared memory*/
  XY[t] = input[t];
  /*****************************************************************/
  /*  YOUR TODO-1 STARTS HERE                                      */
  /*  Implement the Reduction step,                                */
  /*  (the final results is kept in the last element               */
  /*****************************************************************/  
  for(int stride = 1;stride<blockDim.x;stride*=2)
  {
    __syncthreads();
    int index = (t+1)*stride*2-1;
    if(index < blockDim.x) XY[index]+=XY[index-stride];
  }
  /*************************************************************/
  /*    YOUR TODO-1 ENDS HERE                                  */
  /*************************************************************/
  /*************************************************************/
  /*  YOUR TODO-2 STARTS HERE                                  */
  /*  Implement the "post scan" step                           */ 
  /*    to finish the inclusive scan                           */
  /*************************************************************/
  for(int stride=size/4;stride>0;stride/=2)
  {
    __syncthreads();
    int index = (t+1)*stride*2-1;
    if(index+stride < blockDim.x) XY[index+stride] += XY[index];
  }
  /*************************************************************/
  /*    YOUR TODO-2 ENDS HERE                                  */
  /*************************************************************/
  __syncthreads();
  /* write the final output to global memory */
  output[t] = XY[t];
}
/**/
void checkCUDAError(const char *msg);
/**/
int main(int argc, char* argv[])
{   
    int i;
    /**/
    float* h_input, *h_output;
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    /*******************/
    /** READING INPUT **/
    /*******************/
    int size = 0; //dimension of matrices
    /* read the value of size from stdin*/
    scanf("%d", &size);
    /* Allocate host memory */
    h_input = (float*) malloc(sizeof(float)*size);
    h_output = (float*) malloc(sizeof(float)*size);
    /* read input from stdin */
    for(i=0;i<size*size;++i){ scanf("%f", &h_input[i]);}
    /********************/
    /** FINISHED INPUT **/
    /********************/
    /*************************/
    /*  allocate device      */
    /*    memory for A,B,C     */
    /*************************/
    float* d_input, *d_output;
    hipMalloc(&d_input,sizeof(float)*size);
    hipMalloc(&d_output,sizeof(float)*size);
    hipEventRecord(start,0);
    /***********************************/
    /*      copy input data to device  */
    /***********************************/
    hipMemcpy(d_input, h_input, size*sizeof(float), hipMemcpyHostToDevice);
    /*************************************/
    /*       call kernel                 */
    /*       1 block, BLOCK_SIZE threads */
    /*************************************/
    work_efficient_scan<<<1,BLOCK_SIZE>>>(d_input, d_output,size);
    checkCUDAError("Kernel Invoking");
    /**************************/
    /*       copy result back */
    /**************************/
    hipMemcpy(h_output, d_output, sizeof(float)*size, hipMemcpyDeviceToHost);
    /**/
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    fprintf(stderr,"Elapsed time = %f (s)\n",elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    /*******************************************/
    /* Print the final scan result             */
    /*******************************************/
    printf("The final inclusive scan result:\n");
    for(int i=0;i<size;++i)printf("%4.1f  ",h_output[i]);
    /* free device memory */
    hipFree(d_input);
    hipFree(d_output);
    /* free host memory */
    free(h_input);
    free(h_output);
    /**/
    return 0;
}
/*function to test CUDA command*/
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
        hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                        
}
