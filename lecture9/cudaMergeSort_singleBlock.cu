#include <stdio.h>
#include <hip/hip_runtime.h>
/**/
#define SAMPLE_INTERVAL 4 /* pick a sample every 4 elements */
/**/
/* 
   Inline device function, to compute a rank of a "key" in an array "arr" 
   of length "len" (including this key)
*/
static inline __device__ int get_rank_inclusive(int key, int* arr, int len);
/* 
   Inline device function, to compute a rank of a "key" in an array "arr" 
   of length "len" (excluding this key)
*/
static inline __device__ int get_rank_exclusive(int key, int* arr, int len);
/**/
__global__ void pairwise_merge(int* input, int half_size, int* sb_len_left, int* sb_len_right, int sb_num, int* output)
{
  int i, other_rank, output_rank;
  int* left_half = input;
  int* right_half = input + half_size; 
  int* cur_output = output;
  /* A loop through all pair of sub-blocks */
  for(i=0;i<sb_num;++i)
  {
    /***************************************************************/
    /*  YOUR TASK (with 3 TODOs) STARTS HERE                       */
    /*  Perform the pair-wise merging of corresponding sub-blocks  */
    /***************************************************************/
    if(threadIdx.x < sb_len_left[i])
    {
      int key = left_half[threadIdx.x];
      /***************************/
      /* Your TODO-1 starts here */
      /***************************/
      /* use function get_rank_exclusive() to calculate the rank 
        of key in the right_half */
      /* use function get_rank_exclusive() to calculate the rank 
       of key in the left_right*/
      other_rank = get_rank_exclusive(key,right_half,sb_len_right[i]);
      /* calculate the output rank of key */
      output_rank = threadIdx.x + other_rank;
      /* assign key to the correspoding position in the output array*/
      cur_output[output_rank] = key;
    }
    /**/
    
    /********************************************/
    /* Your TODO-2 starts here:                 */
    /* Use the same process as TODO-1           */
    /* to assign the keys in the right half to  */
    /* the output array                         */
    /* hint: use function get_rank_inclusive    */
    /* instead of get_rank_exclusive            */
    /********************************************/
    if(threadIdx.x < sb_len_right[i])
    {
        int key = right_half[threadIdx.x];
         /* use function get_rank_inclusive() to calculate the rank 
        of key in the left_half*/
        other_rank = get_rank_inclusive(key,left_half,sb_len_left[i]);
        /* calculate the output rank of key */
         output_rank = threadIdx.x + other_rank;
        /* assign key to the correspoding position in the output array*/
        cur_output[output_rank] = key;
    }
    /***************************/
    /* Your TODO-2 ends here   */
    /***************************/
    


    /****************************************************/ 
    /* Your TODO-3 starts here:                         */
    /* Update new positions that                        */
    /* left_half, right_half and cur_output point to    */
    /****************************************************/ 
    left_half += sb_len_left[i];
    right_half += sb_len_right[i];
    /**/
    cur_output += sb_len_left[i];
    cur_output += sb_len_right[i];
    /****************************************************/ 
    /* Your TODO-3 ends here                            */
    /****************************************************/ 

    /**********************************************************/
    /*            YOUR TASK ENDS HERE                         */
    /**********************************************************/

    
  } /* end of the loop through all pair of sub-blocks */
  /**/
}/* end of the kernel*/
/**/
void checkCUDAError(const char *msg);
/**/
int main(int argc, char* argv[])
{   
    int i;
    /**/
    int* h_input, *h_output;
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    /*******************/
    /** READING INPUT **/
    /*******************/
    int half_size,size;
    int sb_num; //number of sub-block
    /* read the value of half_size from stdin*/
    scanf("%d", &half_size);
    size = half_size*2;
    /* Allocate host memory */
    h_input = (int*) malloc(sizeof(int)*size);
    h_output = (int*) malloc(sizeof(int)*size);
    /* read input from stdin */
    for(i=0;i<size;++i) scanf("%d", &h_input[i]);
    /* read the value of sb_num */
    scanf("%d", &sb_num);
    int *h_sb_len_left, *h_sb_len_right;
    h_sb_len_left = (int*) malloc(sizeof(int)*sb_num);
    h_sb_len_right = (int*) malloc(sizeof(int)*sb_num);
    for(i=0;i<sb_num;++i) scanf("%d", &h_sb_len_left[i]);
    for(i=0;i<sb_num;++i) scanf("%d", &h_sb_len_right[i]);
    /**/
    /****************************/
    /** FINISHED INPUT READING **/
    /****************************/
    /******************************/
    /*  allocate device memories  */
    /******************************/
    int* d_input, *d_output, *d_sb_len_left, *d_sb_len_right;
    hipMalloc(&d_input,sizeof(int)*size);
    hipMalloc(&d_output,sizeof(int)*size);
    hipMalloc(&d_sb_len_left,sizeof(int)*sb_num);
    hipMalloc(&d_sb_len_right,sizeof(int)*sb_num);
    hipEventRecord(start,0); 
    /***********************************/
    /*      copy input data to device  */
    /***********************************/
    hipMemcpy(d_input, h_input, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sb_len_left, h_sb_len_left, sb_num*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sb_len_right, h_sb_len_right, sb_num*sizeof(int), hipMemcpyHostToDevice);
    /* invoke the kernel, with 1 block, SAMPLE_INTERVAL threads */
    pairwise_merge<<<1,SAMPLE_INTERVAL>>>(d_input,half_size,d_sb_len_left,d_sb_len_right,sb_num,d_output);
    checkCUDAError("kernel invocation\n");
    /* copy the sorted results back to host */
    hipMemcpy(h_output, d_output, sizeof(int)*size, hipMemcpyDeviceToHost);
    /**/
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    fprintf(stderr,"Elapsed time = %f (s)\n",elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    /*******************************************/
    /* Print the final scan result             */
    /*******************************************/
    printf("The sorted array is :\n");
    for(int i=0;i<size;++i) printf("%d  ",h_output[i]);
    printf("\n");
    /* free device memory */
    hipFree(d_sb_len_left);
    hipFree(d_sb_len_right);
    hipFree(d_input);
    hipFree(d_output);
    /* free host memory */
    free(h_input);
    free(h_output);
    free(h_sb_len_left);
    free(h_sb_len_right);
    /**/
    return 0;
}
/*function to test CUDA command*/
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
        hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                        
}
/* 
   Inline device function, to compute a rank of a "key" in an array "arr" 
   of length "len" (including this key)
   Naive implementation. 
   Binary search can be used to implement more efficient function
*/
static inline __device__ int get_rank_inclusive(int key, int* arr, int len)
{
  int rank=0;
  while((rank < len) && (arr[rank]<=key)) ++rank;
  return rank;
}
/* 
   Inline device function, to compute a rank of a "key" in an array "arr" 
   of length "len" (excluding this key)
   Naive implementation. 
   Binary search can be used to implement more efficient function
*/
static inline __device__ int get_rank_exclusive(int key, int* arr, int len)
{
  int rank=0;
  while((rank < len) && (arr[rank]<key)) ++rank;
  return rank;
}
